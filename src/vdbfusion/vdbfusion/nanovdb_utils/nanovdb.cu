#include "hip/hip_runtime.h"
// Copyright Contributors to the OpenVDB Project
// SPDX-License-Identifier: MPL-2.0
#define _USE_MATH_DEFINES

#include <cmath>
#include <chrono>

#include <nanovdb/io/IO.h>
#include <nanovdb/math/Ray.h>
#include <nanovdb/math/HDDA.h>
#include <nanovdb/util/GridBuilder.h>

#include "common.h"

#include <opencv2/opencv.hpp>

#if defined(NANOVDB_USE_CUDA)
#include <nanovdb/cuda/DeviceBuffer.h>
using BufferT = nanovdb::cuda::DeviceBuffer;
#else
using BufferT = nanovdb::HostBuffer;
#endif

void runNanoVDB(nanovdb::GridHandle<BufferT>& handle, nanovdb::GridHandle<BufferT>& label_handle, int width, int height, BufferT& imageBuffer, int index, const std::vector<double> origin)
{ 
    using GridT = nanovdb::FloatGrid;
    using LabelGridT = nanovdb::UInt16Grid;
    using CoordT = nanovdb::Coord;
    using RealT = float;
    using Vec3T = nanovdb::math::Vec3<RealT>;
    using RayT = nanovdb::math::Ray<RealT>;

    auto* h_grid = handle.grid<float>();
    auto* h_label_grid = label_handle.grid<uint16_t>();
    if (!h_grid)
        throw std::runtime_error("GridHandle does not contain a valid host grid");
    if (!h_label_grid)
        throw std::runtime_error("GridHandle does not contain a valid host label grid");

    float* h_outImage = reinterpret_cast<float*>(imageBuffer.data());

#if defined(NANOVDB_USE_CUDA)
    double* d_origin;
    hipMalloc((void**)&d_origin, 3 * sizeof(double));
    hipMemcpy(d_origin, origin.data(), 3 * sizeof(double), hipMemcpyHostToDevice);
#else
    double d_origin[3] = {origin[0], origin[1], origin[2]};
#endif

    float              wBBoxDimZ = (float)h_grid->worldBBox().dim()[2] * 2;
    Vec3T              wBBoxCenter = Vec3T(h_grid->worldBBox().min() + h_grid->worldBBox().dim() * 0.5f);
    nanovdb::CoordBBox treeIndexBbox = h_grid->tree().bbox();
    std::cout << "Bounds: "
              << "[" << treeIndexBbox.min()[0] << "," << treeIndexBbox.min()[1] << "," << treeIndexBbox.min()[2] << "] -> ["
              << treeIndexBbox.max()[0] << "," << treeIndexBbox.max()[1] << "," << treeIndexBbox.max()[2] << "]" << std::endl;

    RayGenOp<Vec3T> rayGenOp(wBBoxDimZ, wBBoxCenter);
    CompositeOp     compositeOp;

    auto renderOp = [width, height, rayGenOp, compositeOp, treeIndexBbox, wBBoxDimZ, d_origin] __hostdev__(int start, int end, float* image, const GridT* grid, const LabelGridT* label_grid) {
        // get an accessor.
        auto acc = grid->tree().getAccessor();
        auto label_acc = label_grid->tree().getAccessor();

        for (int i = start; i < end; ++i) {
            Vec3T rayEye; 
            Vec3T rayDir;
            rayGenOp(i, width, height, rayEye, rayDir);

            // change the ray direction from negative z direction to the positive x direction 
            double rotationMatrix[3][3] = {
                {0, 0, -1},
                {-1, 0, 0},
                {0, 1, 0}
            };

            double x = rayDir[0];
            double y = rayDir[1];
            double z = rayDir[2];
            rayDir[0] = rotationMatrix[0][0] * x + rotationMatrix[0][1] * y + rotationMatrix[0][2] * z;
            rayDir[1] = rotationMatrix[1][0] * x + rotationMatrix[1][1] * y + rotationMatrix[1][2] * z;
            rayDir[2] = rotationMatrix[2][0] * x + rotationMatrix[2][1] * y + rotationMatrix[2][2] * z;

            rayEye[0] = d_origin[0];
            rayEye[1] = d_origin[1];
            rayEye[2] = d_origin[2];

            // generate ray.
            RayT wRay(rayEye, rayDir);

            // transform the ray to the grid's index-space.
            RayT iRay = wRay.worldToIndexF(*grid);

            // intersect...
            float  t0;
            CoordT ijk;
            float  v;

            if (nanovdb::math::ZeroCrossing(iRay, acc, ijk, v, t0)) {
                // write distance to surface. (we assume it is a uniform voxel)
                float wT0 = t0 * float(grid->voxelSize()[0]);
                auto label = label_acc.getValue(ijk);
                compositeOp(image, i, width, height, label, 1.0f);
            } else {
                // write background value.
                compositeOp(image, i, width, height, 0, 0.0f);
 
            }
        }
    };

#if defined(NANOVDB_USE_CUDA)
    auto t5 = std::chrono::high_resolution_clock::now();
    handle.deviceUpload();
    label_handle.deviceUpload();

    auto* d_grid = handle.deviceGrid<float>();
    auto* d_label_grid = label_handle.deviceGrid<uint16_t>();
    if (!d_grid)
        throw std::runtime_error("GridHandle does not contain a valid device grid");
    if (!d_label_grid)
    throw std::runtime_error("GridHandle does not contain a valid device label grid");

    imageBuffer.deviceUpload();
    float* d_outImage = reinterpret_cast<float*>(imageBuffer.deviceData());

    auto t6 = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double, std::milli> elapsed5 = t6 - t5;
    std::cout << "Device Upload took: " << elapsed5.count() << " ms" << std::endl;

    {
        float duration = renderImage(true, renderOp, width, height, d_outImage, d_grid, d_label_grid);
        std::cout << "Duration(NanoVDB-Cuda) = " << duration << " ms" << std::endl;

        auto start3 = std::chrono::high_resolution_clock::now();

        imageBuffer.deviceDownload();

        // std::ostringstream filename;
        // filename << "examples/python/out/pfms/" << "loop_output" << index << ".pfm";

        // Create a cv::Mat of size height x width with 3 channels (CV_32FC3) for storing the image.
        cv::Mat mat(height, width, CV_32FC3);

        auto image = (float*)imageBuffer.data();

        // Populate the cv::Mat with the image data
        for (int i = 0; i < width * height; ++i) {
            int y = height - 1 - (i / width);  // Flip the row index (invert y-axis)
            int x = i % width;

            mat.at<cv::Vec3f>(y, x)[0] = image[2 * width * height + i];  // Blue channel
            mat.at<cv::Vec3f>(y, x)[1] = image[width * height + i];      // Green channel
            mat.at<cv::Vec3f>(y, x)[2] = image[i];                       // Red channel
        }

        mat.convertTo(mat, CV_8UC3, 255);

        cv::imshow("Image", mat);
        cv::waitKey(1);

        // saveImage(filename.str(), width, height, (float*)imageBuffer.data());

        auto end3 = std::chrono::high_resolution_clock::now();
        std::chrono::duration<double, std::milli> elapsed3 = end3 - start3;
        std::cout << "Buffer download and displaying the image took: " << elapsed3.count() << " ms" << std::endl;
    }
#else
    {   
        float duration = renderImage(false, renderOp, width, height, h_outImage, h_grid, h_label_grid);
        std::cout << "Duration(NanoVDB-Host) = " << duration << " ms" << std::endl;

        // std::ostringstream filename;
        // filename << "examples/python/out/pfms/" << "loop_output" << index << ".pfm";
        // saveImage(filename.str(), width, height, (float*)imageBuffer.data());

        auto start3 = std::chrono::high_resolution_clock::now();

        // Create a cv::Mat of size height x width with 3 channels (CV_32FC3) for storing the image.
        cv::Mat mat(height, width, CV_32FC3);

        auto image = (float*)imageBuffer.data();

        // Populate the cv::Mat with the image data
        for (int i = 0; i < width * height; ++i) {
            int y = height - 1 - (i / width);  // Flip the row index (invert y-axis)
            int x = i % width;

            mat.at<cv::Vec3f>(y, x)[0] = image[2 * width * height + i];  // Blue channel
            mat.at<cv::Vec3f>(y, x)[1] = image[width * height + i];      // Green channel
            mat.at<cv::Vec3f>(y, x)[2] = image[i];                       // Red channel
        }

        mat.convertTo(mat, CV_8UC3, 255);

        cv::imshow("Image", mat);
        cv::waitKey(1);

        auto end3 = std::chrono::high_resolution_clock::now();
        std::chrono::duration<double, std::milli> elapsed3 = end3 - start3;
        std::cout << "Displaying the image took: " << elapsed3.count() << " ms" << std::endl;

    }
#endif
}